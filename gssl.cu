#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <iostream>
#include <math.h>
#include "kernels.h"
namespace py = pybind11;

class CudaGSSL {
    public:
        double *a,*c_a;
        double *w,*c_w,*c_wi;
        int n=1;
        std::vector<ssize_t> shape;
        double *c_te,*te;

        CudaGSSL(py::array_t<double> & arr){
            auto buf = arr.request();
            a = (double *)buf.ptr;
            shape = buf.shape;
            for(int i:shape){n*=i;}
            // std::cout<<n<<"#@#@#";
            w = new double [shape[0]*shape[0]];
            te= new double [shape[0]+1];
            hipMalloc( &c_a,n* sizeof(double));
            hipMalloc( &c_w,shape[0]*shape[0]* sizeof(double));
            hipMalloc( &c_wi,shape[0]*shape[0]* sizeof(double));
            for(int k=0;k<shape[0];k++){te[k]=0.0;}
            hipMalloc(&c_te,(shape[0]+1)*sizeof(double));
            hipMemcpy( c_te, te, (shape[0]+1) * sizeof(double),hipMemcpyHostToDevice );
        }
        ~CudaGSSL(){
            delete [] w;
            delete [] te;
            hipFree(c_a);
            hipFree(c_wi);
            hipFree(c_w);
            hipFree(c_te);
            
        }
        py::array_t<double> get_wi(){
            double *wi=new double [shape[0]*shape[0]];
            hipMemcpy(wi,c_wi,shape[0]*shape[0]* sizeof(double),hipMemcpyDeviceToHost);

            py::array_t<double> numpy_array({shape[0],shape[0]}, wi);
            return numpy_array;    
        }

        py::array_t<double> get_w(){
            hipMemcpy(w,c_w,shape[0]*shape[0]* sizeof(double),hipMemcpyDeviceToHost);
            py::array_t<double> numpy_array({shape[0],shape[0]}, w);
            return numpy_array;    
        }

        void gen_w(){


            hipMemcpy( c_a, a, n * sizeof(double),hipMemcpyHostToDevice );

            dim3 block_dim(BLK,BLK) ;
            int threadsPerBlock = 256;
            int blocksPerGrid = (shape[0] + threadsPerBlock - 1) / threadsPerBlock;
            dim3 grids_dim((n*2 + block_dim.x - 1) / block_dim.x, (n*2 + block_dim.y - 1) / block_dim.y);
            weight_matrix_calc<<<grids_dim,block_dim>>>(c_a,c_a,c_w,shape[0],shape[1] );
            hipDeviceSynchronize();
            // GET MEAN OF ELEMENTS
            mean<<<grids_dim,block_dim>>>(c_w, c_te,shape[0]);
            hipDeviceSynchronize();
            //SIGMA^2 TAKEN AS 0.05*MEAN
            final_weight_matrix<<<grids_dim,block_dim>>>(c_w, c_te,shape[0]);
            hipDeviceSynchronize();
            // FIND D^0.5 FOR NORMALIZE
            find_D<<<blocksPerGrid,threadsPerBlock>>>(c_w,c_te,shape[0]);
            hipDeviceSynchronize();
            // D W D
            normalise<<<grids_dim,block_dim>>>(c_w,c_wi,c_te,shape[0]);
            hipDeviceSynchronize();
   
            hipError_t hipError_t = hipGetLastError();
            if (hipError_t != hipSuccess) {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(hipError_t));
                return;
            }

            return;
        }

        py::array_t<double> label_prop(py::array_t<double>  arr){
        // void label_prop(py::array_t<double> & arr){
            auto buf = arr.request();
            double *y,*c_y; //*c_wi,
            
            y = (double *)buf.ptr;
            
            // t_c= new double [shape[0]];
            shape = buf.shape;
            dim3 block_dim(BLK,BLK);
            dim3 grids_dim((2*n + block_dim.x - 1) / block_dim.x, (2*n + block_dim.y - 1) / block_dim.y);

            hipMalloc(&c_y, shape[0]*shape[1]*sizeof(double));
            hipMemcpy(c_y,y , shape[0]*shape[1]*sizeof(double),hipMemcpyHostToDevice );

            // for find inverse 
            for(int k=0;k<shape[0];k++){
                // break;
                // hipDeviceSynchronize();
                // non_zero<<<(shape[0]+1023)/1024,1024>>>(c_w,c_te,k,shape[0]);
                // hipDeviceSynchronize();
                // if (k==0)break;
                // swap<<<(2*shape[0]+(BLK*BLK)-1)/(BLK*BLK),BLK*BLK>>>(c_w, c_wi,c_te,k,shape[0]);

                // hipDeviceSynchronize();
                //TODO: can be more parallelizable
                order_rows<<<1,1>>>(c_w,c_wi,c_te,k,shape[0]);
                hipDeviceSynchronize();
   
                row_opera<<<grids_dim,block_dim>>>(c_w,c_wi,c_te,k,shape[0]);
                hipDeviceSynchronize();

            }

            double *res;
            hipMalloc(&res, shape[0]*shape[1]*sizeof(double));
            mat_mul<<<grids_dim,block_dim>>>(c_wi,c_y,res,shape[0],shape[1]);

            hipMemcpy(y,res,shape[0]*shape[1]* sizeof(double),hipMemcpyDeviceToHost);
            py::array_t<double> numpy_array({shape[0],shape[1]}, y);
            return numpy_array;    
        }
};

PYBIND11_MODULE(gssl,m ) { 
    py::class_<CudaGSSL>(m, "CudaGSSL") 
        .def(py::init<py::array_t<double> &>()) 
        .def("gen_w", &CudaGSSL::gen_w) 
        .def("get_w", &CudaGSSL::get_w)
        .def("get_wi", &CudaGSSL::get_wi)
        .def("label_prop",&CudaGSSL::label_prop);
        

}
//nvcc -arch=compute_60 -code=sm_60 -O3  -shared -std=c++11 -Xcompiler -fPIC $(python3 -m pybind11 --includes) gssl.cu -o gssl$(python3-config --extension-suffix)